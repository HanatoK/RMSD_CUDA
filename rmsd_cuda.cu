#include "hip/hip_runtime.h"
#include "rmsd_cuda.h"
#include "rmsd_cuda_kernel.cuh"
#include <hipsolver.h>
#include <iostream>

bool isDevicePointer(const void* ptr) {
    bool is_device_pointer = true;
    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, ptr);
    if (hipGetLastError() != hipSuccess) {
        std::cout << "here" << std::endl;
        return false;
    }
    if (attributes.devicePointer) {
        is_device_pointer = true;
    } else {
        is_device_pointer = false;
    }
    std::cout << std::boolalpha << is_device_pointer << '\n';
    return is_device_pointer;
}

OptimalRotation::OptimalRotation(const std::vector<AtomPosition>& atom_positions, const std::vector<AtomPosition>& reference_positions):
OptimalRotation(atom_positions.data(), reference_positions.data(), atom_positions.size())
{}

OptimalRotation::OptimalRotation(const AtomPosition* atom_positions, const AtomPosition* reference_positions, const size_t num_atoms): OptimalRotation(num_atoms) {
    hipMemcpyAsync(m_device_reference_positions, reference_positions, m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    hipMemcpyAsync(m_device_atom_positions, atom_positions, m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    calculateOptimalRotationMatrix();
    hipStreamSynchronize(m_stream);
}

OptimalRotation::OptimalRotation(const size_t num_atoms) {
    hipStreamCreate(&m_stream);
    m_num_atoms = num_atoms;
    hipMalloc(&m_device_atom_positions, m_num_atoms * sizeof(AtomPosition));
    hipMalloc(&m_device_reference_positions, m_num_atoms * sizeof(AtomPosition));
    hipMalloc(&m_device_rotation_matrix, 3 * 3 * sizeof(double));
    hipMalloc(&m_device_eigenvalues, 4 * sizeof(double));
    hipMalloc(&m_device_eigenvectors, 4 * 4 * sizeof(double));
    hipMalloc(&devInfo, sizeof(int));
    hipMalloc(&m_center_tmp, sizeof(AtomPosition));
    // initialize the buffer of CUDA eigen solver
    cusolverH = NULL;
    cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    hipsolverSetStream(cusolverH, m_stream);
    hipsolverDnCreateSyevjInfo(&syevj_info);
    hipsolverDnXsyevjSetTolerance(syevj_info, 1e-5);
    hipsolverDnXsyevjSetMaxSweeps(syevj_info, 50);
    lwork = 0;
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
    uplo = HIPBLAS_FILL_MODE_LOWER;
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevj_bufferSize(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, &lwork, syevj_info);
    device_work = nullptr;
    hipMalloc(&device_work, lwork * sizeof(double));
    hipMalloc(&m_device_rmsd, 1 * sizeof(double));
    hipHostMalloc(&m_host_rmsd, 1 * sizeof(double));
    hipMalloc(&d_count, 1 * sizeof(unsigned int));
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
}

void OptimalRotation::updateReference(const std::vector<AtomPosition>& reference_positions) {
    hipMemcpyAsync(m_device_reference_positions, reference_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
}

void OptimalRotation::updateAtoms(const std::vector<AtomPosition>& atom_positions) {
    hipMemcpyAsync(m_device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
}

void OptimalRotation::bringToCenterDevice(AtomPosition* device_atom_positions, const size_t num_atoms) {
    // const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream);
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms);
}

void OptimalRotation::calculateOptimalRotationMatrix() {
    const size_t n_cols = 4;
#ifdef DEBUG
    const size_t n_rows = 4;
#endif
    // const int block_size = 32;
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    // build matrix F
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    hipMemsetAsync(m_device_eigenvectors, 0, 4 * 4 * sizeof(double), m_stream);
    build_matrix_F_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(m_device_atom_positions, m_device_reference_positions, m_device_eigenvectors, m_num_atoms, d_count);

    // device_matrix_F is the eigenvectors after solving
    cusolver_status = hipsolverDnDsyevj(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, device_work, lwork, devInfo, syevj_info);
    // hipStreamSynchronize(m_stream);
    // build the optimal rotation matrix
    build_rotation_matrix_kernel<<<1,1,0,m_stream>>>(m_device_eigenvectors, m_device_rotation_matrix);
    // hipStreamSynchronize(m_stream);
}

// compute the optimal rmsd
double OptimalRotation::minimalRMSD() const {
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream);
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    compute_optimal_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(
        m_device_atom_positions,
        m_device_reference_positions,
        m_device_eigenvalues,
        m_device_rotation_matrix,
        m_device_rmsd, m_num_atoms, d_count);
    hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream);
    hipStreamSynchronize(m_stream);
    return *(m_host_rmsd);
}

// compute the optimal rmsd with respect to a specified frame
double OptimalRotation::minimalRMSD(const std::vector<AtomPosition>& atom_positions) const {
    hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream);
    AtomPosition* device_atom_positions;
    hipMalloc(&device_atom_positions, m_num_atoms * sizeof(AtomPosition));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    // copy data to device
    hipMemcpyAsync(device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    // compute geometric center
    hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream);
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms);
    // we assume the reference frame is already moved to its center of geometry
    // rotate the atoms
    rotate_atoms_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_rotation_matrix, m_num_atoms);
    // compute rmsd directly
    hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    compute_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_reference_positions, m_device_rmsd, m_num_atoms, d_count);
    hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream);
//     hipFree(device_rmsd);
    hipStreamSynchronize(m_stream);
    hipFree(device_atom_positions);
    return *(m_host_rmsd);
}

OptimalRotation::~OptimalRotation() {
    hipFree(m_device_atom_positions);
    hipFree(m_device_reference_positions);
    hipFree(m_device_rotation_matrix);
    hipFree(m_device_eigenvalues);
    hipFree(m_device_eigenvectors);
    hipStreamDestroy(m_stream);
    hipsolverDnDestroySyevjInfo(syevj_info);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipFree(devInfo);
    hipFree(m_center_tmp);
    hipFree(device_work);
    hipFree(m_device_rmsd);
    hipHostFree(m_host_rmsd);
    hipFree(d_count);
    hipDeviceReset();
}
