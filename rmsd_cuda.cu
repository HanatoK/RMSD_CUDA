#include "hip/hip_runtime.h"
#include "rmsd_cuda.h"
#include "rmsd_cuda_kernel.cuh"
#include <hipsolver.h>
#include <iostream>

template <typename T>
void add_copy_node(
  const T* src, T* dst, size_t num_elements,
  hipMemcpyKind kind, hipGraphNode_t& node_out, hipGraph_t& graph,
  std::vector<hipGraphNode_t> dependencies) {
  hipMemcpy3DParms    memcpyParams     = {0};
  memcpyParams.kind     = kind;
  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos   = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr   = make_hipPitchedPtr(
    (void*)src, sizeof(T) * num_elements, num_elements, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos   = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr   = make_hipPitchedPtr(
    (void*)dst, sizeof(T) * num_elements, num_elements, 1);
  memcpyParams.extent   = make_hipExtent(sizeof(T) * num_elements, 1, 1);
  checkCudaErrors(
    hipGraphAddMemcpyNode(
      &node_out, graph, dependencies.data(),
      dependencies.size(), &memcpyParams));
}

template <typename T>
void update_copy_node(
  const T* src, T* dst, size_t num_elements,
  hipMemcpyKind kind, hipGraphNode_t& node,
  hipGraphExec_t& graph_exec) {
  hipMemcpy3DParms    memcpyParams     = {0};
  memcpyParams.kind     = kind;
  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos   = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr   = make_hipPitchedPtr(
    (void*)src, sizeof(T) * num_elements, num_elements, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos   = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr   = make_hipPitchedPtr(
    (void*)dst, sizeof(T) * num_elements, num_elements, 1);
  memcpyParams.extent   = make_hipExtent(sizeof(T) * num_elements, 1, 1);
  checkCudaErrors(hipGraphExecMemcpyNodeSetParams(
    graph_exec, node,
    &memcpyParams));
}

template <typename T>
void add_clear_array_node(
  T* dst, const size_t num_elements,
  hipGraphNode_t& node_out, hipGraph_t& graph,
  const std::vector<hipGraphNode_t>& dependencies) {
  // size_t elementSize, width;
  const size_t sizeofT = sizeof(T);
  /**< Size of each element in bytes. Must be 1, 2, or 4. */
  const size_t elementSize =
    (sizeofT % 4 == 0) ? 4 :
    ((sizeofT % 2 == 0) ? 2 : 1);
  const size_t width = num_elements * (sizeofT / elementSize);
  hipMemsetParams memsetParams = {0};
  memsetParams.dst         = (void*)dst;
  memsetParams.value       = 0;
  memsetParams.elementSize = elementSize;
  memsetParams.width       = width;
  memsetParams.height      = 1;
  checkCudaErrors(hipGraphAddMemsetNode(
    &node_out, graph, dependencies.data(),
    dependencies.size(), &memsetParams));
}

bool isDevicePointer(const void* ptr) {
    bool is_device_pointer = true;
    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, ptr);
    if (hipGetLastError() != hipSuccess) {
        std::cout << "here" << std::endl;
        return false;
    }
    if (attributes.devicePointer) {
        is_device_pointer = true;
    } else {
        is_device_pointer = false;
    }
    std::cout << std::boolalpha << is_device_pointer << '\n';
    return is_device_pointer;
}

OptimalRotation::OptimalRotation(const size_t num_atoms)
#if defined (USE_CUDA_GRAPH)
    : graphCreated(false)
#endif
{
    checkCudaErrors(hipStreamCreate(&m_stream));
#if defined(USE_CUDA_GRAPH)
    std::memset(&gpu_nodes, 0, sizeof(gpu_nodes));
    checkCudaErrors(hipGraphCreate(&m_graph, 0));
    m_instance = NULL;
#endif
    m_num_atoms = num_atoms;
    std::cout << "Number of atoms: " << num_atoms << std::endl;
    checkCudaErrors(hipMalloc(&m_device_atom_positions, m_num_atoms * sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_device_reference_positions, m_num_atoms * sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_device_rotation_matrix, 3 * 3 * sizeof(double)));
    checkCudaErrors(hipMalloc(&m_device_eigenvalues, 4 * sizeof(double)));
    checkCudaErrors(hipMalloc(&m_device_eigenvectors, 4 * 4 * sizeof(double)));
#if defined (USE_CUDA_GRAPH)
    checkCudaErrors(hipMalloc(&m_center_tmp_ref, sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_center_tmp_pos, sizeof(AtomPosition)));
#else
    checkCudaErrors(hipMalloc(&m_center_tmp, sizeof(AtomPosition)));
#endif // USE_CUDA_GRAPH
    checkCudaErrors(hipMalloc(&m_device_rmsd, 1 * sizeof(double)));
#if !defined (USE_NR)
    // initialize the buffer of CUDA eigen solver
    cusolverH = NULL;
    hipMalloc(&devInfo, sizeof(int));
    cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    hipsolverSetStream(cusolverH, m_stream);
    hipsolverDnCreateSyevjInfo(&syevj_info);
    hipsolverDnXsyevjSetTolerance(syevj_info, 1e-5);
    hipsolverDnXsyevjSetMaxSweeps(syevj_info, 50);
    lwork = 0;
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
    uplo = HIPBLAS_FILL_MODE_LOWER;
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevj_bufferSize(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, &lwork, syevj_info);
    device_work = nullptr;
    hipMalloc(&device_work, lwork * sizeof(double));
#endif // !defined (USE_NR)
// #if defined(USE_CUDA_GRAPH)
    checkCudaErrors(hipHostMalloc(&m_host_rmsd, 1 * sizeof(double)));
#if defined (USE_CUDA_GRAPH)
    checkCudaErrors(hipMalloc(&d_count_ref, 1 * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc(&d_count_pos, 1 * sizeof(unsigned int)));
#endif
    checkCudaErrors(hipMalloc(&d_count, 1 * sizeof(unsigned int)));
    // hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    mEventAttrib.version = NVTX_VERSION;
    mEventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    mEventAttrib.colorType = NVTX_COLOR_ARGB;
    mEventAttrib.color = 0xFF880000;
    mEventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
#if defined (USE_NR)
    mEventAttrib.message.ascii = "NR";
#else
    mEventAttrib.message.ascii = "cuSolver";
#endif
#if defined (USE_NR)
    checkCudaErrors(hipHostMalloc(&max_reached, sizeof(int)));
    max_reached[0] = 0;
#endif
}

void OptimalRotation::updateReference(const host_vector<AtomPosition>& reference_positions) {
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        add_copy_node(
            reference_positions.data(),
            m_device_reference_positions,
            m_num_atoms, hipMemcpyHostToDevice,
            gpu_nodes.updateReferenceNode, m_graph, {});
        bringToCenterDevice(
            m_device_reference_positions, m_num_atoms,
            m_center_tmp_ref, d_count_ref,
            {gpu_nodes.updateReferenceNode},
            gpu_nodes.centerReferenceNode);
    } else {
        update_copy_node(
            reference_positions.data(),
            m_device_reference_positions,
            m_num_atoms, hipMemcpyHostToDevice,
            gpu_nodes.updateReferenceNode, m_instance);
    }
#else
    checkCudaErrors(hipMemcpyAsync(m_device_reference_positions, reference_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
#endif
}

void OptimalRotation::updateAtoms(const host_vector<AtomPosition>& atom_positions) {
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        add_copy_node(
            atom_positions.data(),
            m_device_atom_positions,
            m_num_atoms, hipMemcpyHostToDevice,
            gpu_nodes.updateAtomsNode, m_graph, {});
        bringToCenterDevice(
            m_device_atom_positions, m_num_atoms,
            m_center_tmp_pos, d_count_pos,
            {gpu_nodes.updateAtomsNode},
            gpu_nodes.centerAtomsNode);
    } else {
        update_copy_node(
            atom_positions.data(),
            m_device_atom_positions,
            m_num_atoms, hipMemcpyHostToDevice,
            gpu_nodes.updateAtomsNode, m_instance);
    }
#else
    checkCudaErrors(hipMemcpyAsync(m_device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
#endif
}

void OptimalRotation::bringToCenterDevice(AtomPosition* device_atom_positions, const size_t num_atoms
#if defined(USE_CUDA_GRAPH)
, AtomPosition* center_out,
unsigned int* counter,
std::vector<hipGraphNode_t> dependencies,
hipGraphNode_t& last_node
#endif
) {
    // const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        hipGraphNode_t counterSetNode;
        hipGraphNode_t centerSetNode;
        hipGraphNode_t getCenterKernelNode;
        hipGraphNode_t moveToCenterKernelNode;
        // hipMemsetParams memsetParams = {0};
        // memsetParams.dst            = d_count;
        // memsetParams.value          = 0;
        // memsetParams.elementSize    = 1 * sizeof(unsigned int);
        // memsetParams.width          = 1;
        // memsetParams.height         = 1;
        // checkCudaErrors(hipGraphAddMemsetNode(&counterSetNode, m_graph, dependencies.data(), dependencies.size(), &memsetParams));
        // memsetParams.dst            = m_center_tmp;
        // memsetParams.elementSize    = sizeof(float);
        // memsetParams.width          = 1 * sizeof(AtomPosition) / memsetParams.elementSize;
        // checkCudaErrors(hipGraphAddMemsetNode(&centerSetNode, m_graph, dependencies.data(), dependencies.size(), &memsetParams));
        add_clear_array_node(counter, 1, counterSetNode, m_graph, {});
        add_clear_array_node(center_out, 1, centerSetNode, m_graph, {});
        // Run kernels
        hipKernelNodeParams kernelNodeParams = {0};
        // hipGraphNode_t dependencies[] = {
        //     counterSetNode,
        //     centerSetNode};
        dependencies.push_back(counterSetNode);
        dependencies.push_back(centerSetNode);
        const void *getCenterKernelArgs[] =
            {&device_atom_positions, &center_out, &num_atoms, &counter};
        kernelNodeParams.func           = (void*)get_center_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = const_cast<void**>(getCenterKernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&getCenterKernelNode, m_graph, dependencies.data(), dependencies.size(), &kernelNodeParams));
        // last_node = getCenterKernelNode;
        const void* moveAtomToCenterKernelArgs[] =
            {&device_atom_positions, &center_out, &num_atoms};
        kernelNodeParams.func           = (void*)move_atom_to_center_kernel;
        kernelNodeParams.kernelParams   = const_cast<void**>(moveAtomToCenterKernelArgs);
        checkCudaErrors(hipGraphAddKernelNode(&moveToCenterKernelNode, m_graph, &getCenterKernelNode, 1, &kernelNodeParams));
        last_node = moveToCenterKernelNode;
    }
#else
    checkCudaErrors(hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms);
#endif
}

void OptimalRotation::calculateOptimalRotationMatrix() {
#ifdef DEBUG
    const size_t n_rows = 4;
#endif
    // const int block_size = 32;
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Memsets
        hipGraphNode_t counterSetNode, eigenVectorsSetNode;
        add_clear_array_node(
            d_count, 1, counterSetNode, m_graph,
            {});
        add_clear_array_node(
            m_device_eigenvectors, 4*4,
            eigenVectorsSetNode, m_graph,
            {});
        // last_node = eigenVectorsSetNode;
        // build matrix F
        std::vector<hipGraphNode_t> dependencies =
            {counterSetNode,
             eigenVectorsSetNode,
            gpu_nodes.centerReferenceNode,
            gpu_nodes.centerAtomsNode
            };
        hipKernelNodeParams kernelNodeParams = {0};
        const void *buildMatrixFKernelArgs[] =
            {&m_device_atom_positions, &m_device_reference_positions, &m_device_eigenvectors, &m_num_atoms, &d_count};
        kernelNodeParams.func           = (void*)build_matrix_F_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   =
            const_cast<void**>(buildMatrixFKernelArgs);
        kernelNodeParams.extra          = NULL;
        hipGraphAddKernelNode(&gpu_nodes.buildMatrixFNode, m_graph, dependencies.data(), dependencies.size(), &kernelNodeParams);
        // last_node = buildMatrixFNode;
    }
#else
    // build matrix F
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    checkCudaErrors(hipMemsetAsync(m_device_eigenvectors, 0, 4 * 4 * sizeof(double), m_stream));
    build_matrix_F_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(m_device_atom_positions, m_device_reference_positions, m_device_eigenvectors, m_num_atoms, d_count);

    nvtxRangePushEx(&mEventAttrib);
#endif // defined(USE_CUDA_GRAPH)
#if defined(USE_NR)
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Jacobi node
        hipKernelNodeParams kernelNodeParams = {0};
        const void *kernelArgs[] = {
            &m_device_eigenvectors, &m_device_eigenvalues, &max_reached};
        kernelNodeParams.func           = (void*)jacobi_4x4;
        kernelNodeParams.gridDim        = dim3(1, 1, 1);
        kernelNodeParams.blockDim       = dim3(2, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   =
            const_cast<void**>(kernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&gpu_nodes.jacobi4x4Node, m_graph, &gpu_nodes.buildMatrixFNode, 1, &kernelNodeParams));
    }
    // getLastCudaError("Line 312\n");
#else
    jacobi_4x4<<<1,2,0,m_stream>>>(m_device_eigenvectors, m_device_eigenvalues, max_reached);
    if (max_reached[0] > 0) {
        std::cerr << "Maximum number of iterations reached!\n";
    }
#endif // defined(USE_CUDA_GRAPH)
#else
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevj(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, device_work, lwork, devInfo, syevj_info);
#endif
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Build rotation matrix node
        hipKernelNodeParams kernelNodeParams = {0};
        size_t max_eigenvalue_index = 3;
        void *kernelArgs[] = {
            &m_device_eigenvectors, &m_device_rotation_matrix, &max_eigenvalue_index};
        kernelNodeParams.func           = (void*)build_rotation_matrix_kernel;
        kernelNodeParams.gridDim        = dim3(1, 1, 1);
        kernelNodeParams.blockDim       = dim3(1, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = kernelArgs;
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&gpu_nodes.buildRotationMatrixKernelNode, m_graph, &gpu_nodes.jacobi4x4Node, 1, &kernelNodeParams));
    }
#else
    nvtxRangePop();
    // hipStreamSynchronize(m_stream);
    // build the optimal rotation matrix
    build_rotation_matrix_kernel<<<1,1,0,m_stream>>>(m_device_eigenvectors, m_device_rotation_matrix);
    // hipStreamSynchronize(m_stream);
#endif
}

// compute the optimal rmsd
#if defined (USE_CUDA_GRAPH)
double OptimalRotation::minimalRMSD() {
#else
double OptimalRotation::minimalRMSD() const {
#endif
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        hipGraphNode_t last_node = gpu_nodes.buildRotationMatrixKernelNode;
        // Memsets
        hipGraphNode_t counterSetNode, deviceRMSDSetNode;
        hipMemsetParams memsetParams = {0};
        memsetParams.dst            = d_count;
        memsetParams.value          = 0;
        memsetParams.elementSize    = 1 * sizeof(unsigned int);
        memsetParams.width          = 1;
        memsetParams.height         = 1;
        checkCudaErrors(hipGraphAddMemsetNode(
            &counterSetNode, m_graph, &last_node, 1, &memsetParams));
        // last_node = counterSetNode;
        memsetParams.dst            = m_device_rmsd;
        memsetParams.elementSize    = sizeof(float);
        memsetParams.width          = sizeof(double) / memsetParams.elementSize;
        checkCudaErrors(hipGraphAddMemsetNode(
            &deviceRMSDSetNode, m_graph, &last_node, 1, &memsetParams));
        // last_node = deviceRMSDSetNode;
        hipGraphNode_t RMSDDependencies[] = {counterSetNode, deviceRMSDSetNode};
        // Compute RMSD
        hipGraphNode_t RMSDKernelNode;
        hipKernelNodeParams kernelNodeParams = {0};
        const size_t max_eigenvalue_index = 3;
        const void *kernelArgs[] = {
            &m_device_atom_positions,
            &m_device_reference_positions,
            &m_device_eigenvalues,
            &m_device_rotation_matrix,
            &m_device_rmsd, &m_num_atoms, &d_count,
            &max_eigenvalue_index};
        kernelNodeParams.func           =
            (void*)compute_optimal_rmsd_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = const_cast<void**>(kernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(
            &RMSDKernelNode, m_graph, RMSDDependencies, 2, &kernelNodeParams));
        // last_node = RMSDKernelNode;
        // Instantiate graph
        checkCudaErrors(hipGraphInstantiate(&m_instance, m_graph, NULL, NULL, 0));
        graphCreated = true;
    }
    static int iter = 0;
    if (iter /*% 2*/ == 0) {
        hipGraphDebugDotFlags dotFlags = hipGraphDebugDotFlagsVerbose;
        const std::string filename = "graph_" + std::to_string(iter) + ".dot";
        checkCudaErrors(hipGraphDebugDotPrint(m_graph, filename.c_str(), dotFlags));
    }
    iter++;
    // Run graph
    checkCudaErrors(hipGraphLaunch(m_instance, m_stream));
#else
    checkCudaErrors(hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    compute_optimal_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(
        m_device_atom_positions,
        m_device_reference_positions,
        m_device_eigenvalues,
        m_device_rotation_matrix,
        m_device_rmsd, m_num_atoms, d_count);
#endif
    checkCudaErrors(hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream));
    checkCudaErrors(hipStreamSynchronize(m_stream));
    return *(m_host_rmsd);
}

#if defined (USE_CUDA_GRAPH)
void OptimalRotation::resetGraph() {
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipGraphExecDestroy(m_instance));
    checkCudaErrors(hipGraphDestroy(m_graph));
    checkCudaErrors(hipStreamDestroy(m_stream));
    graphCreated = false;
    // last_node = NULL;
    // Recreate graph
    checkCudaErrors(hipStreamCreate(&m_stream));
    checkCudaErrors(hipGraphCreate(&m_graph, 0));
}
#endif // USE_CUDA_GRAPH

#if defined (USE_CUDA_GRAPH)
#else
// compute the optimal rmsd with respect to a specified frame
double OptimalRotation::minimalRMSD(const host_vector<AtomPosition>& atom_positions) const {
    checkCudaErrors(hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream));
    AtomPosition* device_atom_positions;
    checkCudaErrors(hipMalloc(&device_atom_positions, m_num_atoms * sizeof(AtomPosition)));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    // copy data to device
    checkCudaErrors(hipMemcpyAsync(device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    // compute geometric center
    checkCudaErrors(hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms);
    // we assume the reference frame is already moved to its center of geometry
    // rotate the atoms
    rotate_atoms_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_rotation_matrix, m_num_atoms);
    // compute rmsd directly
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    compute_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_reference_positions, m_device_rmsd, m_num_atoms, d_count);
    checkCudaErrors(hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream));
//     hipFree(device_rmsd);
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipFree(device_atom_positions));
    return *(m_host_rmsd);
}
#endif

OptimalRotation::~OptimalRotation() {
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipFree(m_device_atom_positions));
    checkCudaErrors(hipFree(m_device_reference_positions));
    checkCudaErrors(hipFree(m_device_rotation_matrix));
    checkCudaErrors(hipFree(m_device_eigenvalues));
    checkCudaErrors(hipFree(m_device_eigenvectors));
    checkCudaErrors(hipStreamDestroy(m_stream));
#if !defined (USE_NR)
    hipsolverDnDestroySyevjInfo(syevj_info);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    checkCudaErrors(hipFree(devInfo));
    checkCudaErrors(hipFree(device_work));
#endif
#if defined (USE_CUDA_GRAPH)
    checkCudaErrors(hipFree(m_center_tmp_ref));
    checkCudaErrors(hipFree(m_center_tmp_pos));
    checkCudaErrors(hipFree(d_count_ref));
    checkCudaErrors(hipFree(d_count_pos));
#else
    checkCudaErrors(hipFree(m_center_tmp));
#endif
    checkCudaErrors(hipFree(m_device_rmsd));
    checkCudaErrors(hipHostFree(m_host_rmsd));
    checkCudaErrors(hipFree(d_count));
#if defined(USE_NR)
    checkCudaErrors(hipHostFree(max_reached));
#endif
#if defined (USE_CUDA_GRAPH)
    if (m_graph) checkCudaErrors(hipGraphDestroy(m_graph));
    if (m_instance) checkCudaErrors(hipGraphExecDestroy(m_instance));
#endif
}
