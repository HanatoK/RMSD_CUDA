#include "hip/hip_runtime.h"
#include "rmsd_cuda.h"
#include "rmsd_cuda_kernel.h"
#include <hipsolver.h>
#include <iostream>

bool isDevicePointer(const void* ptr) {
    bool is_device_pointer = true;
    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, ptr);
    if (hipGetLastError() != hipSuccess) {
        std::cout << "here" << std::endl;
        return false;
    }
    if (attributes.devicePointer) {
        is_device_pointer = true;
    } else {
        is_device_pointer = false;
    }
    std::cout << std::boolalpha << is_device_pointer << '\n';
    return is_device_pointer;
}

OptimalRotation::OptimalRotation(const std::vector<AtomPosition>& atom_positions, const std::vector<AtomPosition>& reference_positions):
OptimalRotation(atom_positions.data(), reference_positions.data(), atom_positions.size())
{}

OptimalRotation::OptimalRotation(const AtomPosition* atom_positions, const AtomPosition* reference_positions, const size_t num_atoms): OptimalRotation(num_atoms) {
    hipMemcpyAsync(m_device_reference_positions, reference_positions, m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    hipMemcpyAsync(m_device_atom_positions, atom_positions, m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    calculateOptimalRotationMatrix();
    hipStreamSynchronize(m_stream);
}

OptimalRotation::OptimalRotation(const size_t num_atoms) {
    hipStreamCreate(&m_stream);
    m_num_atoms = num_atoms;
    hipMalloc(&m_device_atom_positions, m_num_atoms * sizeof(AtomPosition));
    hipMalloc(&m_device_reference_positions, m_num_atoms * sizeof(AtomPosition));
    hipMalloc(&m_device_rotation_matrix, 3 * 3 * sizeof(double));
    hipMalloc(&m_device_eigenvalues, 4 * sizeof(double));
    hipMalloc(&m_device_eigenvectors, 4 * 4 * sizeof(double));
    hipMalloc(&devInfo, sizeof(int));
    hipMalloc(&m_center_tmp, 3 * sizeof(AtomPosition));
    // initialize the buffer of CUDA eigen solver
    cusolverH = NULL;
    cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    hipsolverSetStream(cusolverH, m_stream);
    lwork = 0;
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
    uplo = HIPBLAS_FILL_MODE_LOWER;
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, &lwork);
    device_work = nullptr;
    hipMalloc(&device_work, lwork * sizeof(double));
    hipMalloc(&m_device_rmsd, 1 * sizeof(double));
    hipHostMalloc(&m_host_rmsd, 1 * sizeof(double));
}

void OptimalRotation::updateReference(const std::vector<AtomPosition>& reference_positions) {
    hipMemcpyAsync(m_device_reference_positions, reference_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    hipStreamSynchronize(m_stream);
}

void OptimalRotation::updateAtoms(const std::vector<AtomPosition>& atom_positions) {
    hipMemcpyAsync(m_device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    hipStreamSynchronize(m_stream);
}

void OptimalRotation::bringToCenterDevice(AtomPosition* device_atom_positions, const size_t num_atoms) {
    const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    hipMemsetAsync(m_center_tmp, 0, 3 * sizeof(double3), m_stream);
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms);
}

void OptimalRotation::calculateOptimalRotationMatrix() {
    const size_t n_cols = 4;
#ifdef DEBUG
    const size_t n_rows = 4;
#endif
    const int block_size = 32;
    const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    // build matrix F
    build_matrix_F_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(m_device_atom_positions, m_device_reference_positions, m_device_eigenvectors, m_num_atoms);

    // device_matrix_F is the eigenvectors after solving
    cusolver_status = hipsolverDnDsyevd(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, device_work, lwork, devInfo);

#ifdef DEBUG
    {
        hipStreamSynchronize(m_stream);
        double host_eigenvalues[n_cols];
        double host_eigenvectors[n_cols * n_rows];
        hipMemcpy(host_eigenvalues, m_device_eigenvalues, n_cols * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(host_eigenvectors, m_device_eigenvectors, n_cols * n_rows * sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "Eigenvalues: " << '\n';
        for (size_t i = 0; i < n_cols; ++i) {
            std::cout << host_eigenvalues[i] << " ";
        }
        std::cout << '\n';
        std::cout << "Eigenvectors (in collumn vectors): " << '\n';
        for (size_t i = 0; i < n_rows; ++i) {
            for (size_t j = 0; j < n_cols; ++j) {
                const double elem = host_eigenvectors[i + j * n_cols];
                std::cout << elem << " ";
            }
            std::cout << '\n';
        }
    }
#endif
    // build the optimal rotation matrix
    build_rotation_matrix_kernel<<<1,1,0,m_stream>>>(m_device_eigenvectors, m_device_rotation_matrix);
    hipStreamSynchronize(m_stream);
#ifdef DEBUG
    {
        double host_eigenvalues[n_cols];
        double host_eigenvectors[n_cols * n_rows];
        hipMemcpy(host_eigenvalues, m_device_eigenvalues, n_cols * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(host_eigenvectors, m_device_eigenvectors, n_cols * n_rows * sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "After copy Eigenvalues: " << '\n';
        for (size_t i = 0; i < n_cols; ++i) {
            std::cout << host_eigenvalues[i] << " ";
        }
        std::cout << '\n';
        std::cout << "After copy Eigenvectors (in collumn vectors): " << '\n';
        for (size_t i = 0; i < n_rows; ++i) {
            for (size_t j = 0; j < n_cols; ++j) {
                const double elem = host_eigenvectors[i + j * n_cols];
                std::cout << elem << " ";
            }
            std::cout << '\n';
        }
    }
#endif
}

// compute the optimal rmsd
double OptimalRotation::minimalRMSD() const {
//     double rmsd;
//     double* device_rmsd;
//     hipMalloc(&device_rmsd, 1 * sizeof(double));
    hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream);
    const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    compute_optimal_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(m_device_atom_positions, m_device_reference_positions, m_device_eigenvalues, m_device_rmsd, m_num_atoms);
    hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream);
//     hipFree(device_rmsd);
    hipStreamSynchronize(m_stream);
    return *(m_host_rmsd);
}

// compute the optimal rmsd with respect to a specified frame
double OptimalRotation::minimalRMSD(const std::vector<AtomPosition>& atom_positions) const {
//     double rmsd;
//     double* device_rmsd;
//     hipMalloc(&device_rmsd, 1 * sizeof(double));
    hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream);
    AtomPosition* device_atom_positions;
    hipMalloc(&device_atom_positions, m_num_atoms * sizeof(AtomPosition));
    const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    // copy data to device
    hipMemcpyAsync(device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream);
    // compute geometric center
    hipMemsetAsync(m_center_tmp, 0, 3 * sizeof(double3), m_stream);
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms);
    // we assume the reference frame is already moved to its center of geometry
    // rotate the atoms
    rotate_atoms_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_rotation_matrix, m_num_atoms);
    // compute rmsd directly
    compute_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_reference_positions, m_device_rmsd, m_num_atoms);
    hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream);
//     hipFree(device_rmsd);
    hipStreamSynchronize(m_stream);
    hipFree(device_atom_positions);
    return *(m_host_rmsd);
}

OptimalRotation::~OptimalRotation() {
    hipFree(m_device_atom_positions);
    hipFree(m_device_reference_positions);
    hipFree(m_device_rotation_matrix);
    hipFree(m_device_eigenvalues);
    hipFree(m_device_eigenvectors);
    hipStreamDestroy(m_stream);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipFree(devInfo);
    hipFree(m_center_tmp);
    hipFree(device_work);
    hipFree(m_device_rmsd);
    hipHostFree(m_host_rmsd);
    hipDeviceReset();
}
