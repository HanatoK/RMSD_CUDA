#include "hip/hip_runtime.h"
#include "rmsd_cuda.h"
#include "rmsd_cuda_kernel.cuh"
#include <hipsolver.h>
#include <iostream>

bool isDevicePointer(const void* ptr) {
    bool is_device_pointer = true;
    hipPointerAttribute_t attributes;
    hipPointerGetAttributes(&attributes, ptr);
    if (hipGetLastError() != hipSuccess) {
        std::cout << "here" << std::endl;
        return false;
    }
    if (attributes.devicePointer) {
        is_device_pointer = true;
    } else {
        is_device_pointer = false;
    }
    std::cout << std::boolalpha << is_device_pointer << '\n';
    return is_device_pointer;
}

OptimalRotation::OptimalRotation(const size_t num_atoms)
#if defined (USE_CUDA_GRAPH)
    : last_node(NULL), graphCreated(false)
#endif
{
    checkCudaErrors(hipStreamCreate(&m_stream));
#if defined(USE_CUDA_GRAPH)
    checkCudaErrors(hipGraphCreate(&m_graph, 0));
    m_instance = NULL;
#endif
    m_num_atoms = num_atoms;
    std::cout << "Number of atoms: " << num_atoms << std::endl;
    checkCudaErrors(hipMalloc(&m_device_atom_positions, m_num_atoms * sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_device_reference_positions, m_num_atoms * sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_device_rotation_matrix, 3 * 3 * sizeof(double)));
    checkCudaErrors(hipMalloc(&m_device_eigenvalues, 4 * sizeof(double)));
    checkCudaErrors(hipMalloc(&m_device_eigenvectors, 4 * 4 * sizeof(double)));
    checkCudaErrors(hipMalloc(&m_center_tmp, sizeof(AtomPosition)));
    checkCudaErrors(hipMalloc(&m_device_rmsd, 1 * sizeof(double)));
#if !defined (USE_NR)
    // initialize the buffer of CUDA eigen solver
    cusolverH = NULL;
    hipMalloc(&devInfo, sizeof(int));
    cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    cusolver_status = hipsolverDnCreate(&cusolverH);
    hipsolverSetStream(cusolverH, m_stream);
    hipsolverDnCreateSyevjInfo(&syevj_info);
    hipsolverDnXsyevjSetTolerance(syevj_info, 1e-5);
    hipsolverDnXsyevjSetMaxSweeps(syevj_info, 50);
    lwork = 0;
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
    uplo = HIPBLAS_FILL_MODE_LOWER;
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevj_bufferSize(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, &lwork, syevj_info);
    device_work = nullptr;
    hipMalloc(&device_work, lwork * sizeof(double));
#endif // !defined (USE_NR)
// #if defined(USE_CUDA_GRAPH)
    checkCudaErrors(hipHostMalloc(&m_host_rmsd, 1 * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_count, 1 * sizeof(unsigned int)));
    // hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream);
    mEventAttrib.version = NVTX_VERSION;
    mEventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    mEventAttrib.colorType = NVTX_COLOR_ARGB;
    mEventAttrib.color = 0xFF880000;
    mEventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
#if defined (USE_NR)
    mEventAttrib.message.ascii = "NR";
#else
    mEventAttrib.message.ascii = "cuSolver";
#endif
#if defined (USE_NR)
    checkCudaErrors(hipHostMalloc(&max_reached, sizeof(int)));
    max_reached[0] = 0;
#endif
}

void OptimalRotation::updateReference(const host_vector<AtomPosition>& reference_positions) {
    checkCudaErrors(hipMemcpyAsync(m_device_reference_positions, reference_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    bringToCenterDevice(m_device_reference_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
}

void OptimalRotation::updateAtoms(const host_vector<AtomPosition>& atom_positions) {
    checkCudaErrors(hipMemcpyAsync(m_device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    bringToCenterDevice(m_device_atom_positions, m_num_atoms);
    // hipStreamSynchronize(m_stream);
}

void OptimalRotation::bringToCenterDevice(AtomPosition* device_atom_positions, const size_t num_atoms) {
    // const int num_blocks = int(std::ceil(double(m_num_atoms) / block_size));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        hipGraphNode_t counterSetNode, centerSetNode, getCenterKernelNode, moveToCenterKernel;
        hipMemsetParams memsetParams = {0};
        memsetParams.dst            = d_count;
        memsetParams.value          = 0;
        memsetParams.elementSize    = 1 * sizeof(unsigned int);
        memsetParams.width          = 1;
        memsetParams.height         = 1;
        if (last_node == nullptr) {
            checkCudaErrors(hipGraphAddMemsetNode(&counterSetNode, m_graph, NULL, 0, &memsetParams));
        } else {
            checkCudaErrors(hipGraphAddMemsetNode(&counterSetNode, m_graph, &last_node, 1, &memsetParams));
        }
        // checkCudaErrors(hipGraphAddMemsetNode(&counterSetNode, m_graph, NULL, 0, &memsetParams));
        // last_node = counterSetNode;
        memsetParams.dst            = m_center_tmp;
        memsetParams.elementSize    = sizeof(float);
        memsetParams.width          = 1 * sizeof(AtomPosition) / memsetParams.elementSize;
        if (last_node == nullptr) {
            checkCudaErrors(hipGraphAddMemsetNode(&centerSetNode, m_graph, NULL, 0, &memsetParams));
        } else {
            checkCudaErrors(hipGraphAddMemsetNode(&centerSetNode, m_graph, &last_node, 1, &memsetParams));
        }
        // last_node = centerSetNode;
        // Run kernels
        hipKernelNodeParams kernelNodeParams = {0};
        hipGraphNode_t dependencies[] = {counterSetNode, centerSetNode};
        const void *getCenterKernelArgs[] =
            {&device_atom_positions, &m_center_tmp, &num_atoms, &d_count};
        kernelNodeParams.func           = (void*)get_center_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = const_cast<void**>(getCenterKernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&getCenterKernelNode, m_graph, dependencies, 2, &kernelNodeParams));
        last_node = getCenterKernelNode;
        const void* moveAtomToCenterKernelArgs[] =
            {&device_atom_positions, &m_center_tmp, &num_atoms};
        kernelNodeParams.func           = (void*)move_atom_to_center_kernel;
        kernelNodeParams.kernelParams   = const_cast<void**>(moveAtomToCenterKernelArgs);
        checkCudaErrors(hipGraphAddKernelNode(&moveToCenterKernel, m_graph, &last_node, 1, &kernelNodeParams));
        last_node = moveToCenterKernel;
    }
#else
    checkCudaErrors(hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, num_atoms);
#endif
}

void OptimalRotation::calculateOptimalRotationMatrix() {
#ifdef DEBUG
    const size_t n_rows = 4;
#endif
    // const int block_size = 32;
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Memsets
        hipGraphNode_t counterSetNode, eigenVectorsSetNode;
        hipMemsetParams memsetParams = {0};
        memsetParams.dst            = d_count;
        memsetParams.value          = 0;
        memsetParams.elementSize    = 1 * sizeof(unsigned int);
        memsetParams.width          = 1;
        memsetParams.height         = 1;
        hipGraphAddMemsetNode(
            &counterSetNode, m_graph, &last_node, 1, &memsetParams);
        // last_node = counterSetNode;
        memsetParams.dst            = m_device_eigenvectors;
        memsetParams.elementSize    = sizeof(float);
        memsetParams.width          = 4 * 4 * sizeof(double) / memsetParams.elementSize;
        hipGraphAddMemsetNode(
            &eigenVectorsSetNode, m_graph, &last_node, 1, &memsetParams);
        // last_node = eigenVectorsSetNode;
        // build matrix F
        hipGraphNode_t dependencies[] = {counterSetNode, eigenVectorsSetNode};
        hipGraphNode_t buildMatrixFNode;
        hipKernelNodeParams kernelNodeParams = {0};
        const void *buildMatrixFKernelArgs[] =
            {&m_device_atom_positions, &m_device_reference_positions, &m_device_eigenvectors, &m_num_atoms, &d_count};
        kernelNodeParams.func           = (void*)build_matrix_F_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   =
            const_cast<void**>(buildMatrixFKernelArgs);
        kernelNodeParams.extra          = NULL;
        hipGraphAddKernelNode(&buildMatrixFNode, m_graph, dependencies, 2, &kernelNodeParams);
        last_node = buildMatrixFNode;
    }
#else
    // build matrix F
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    checkCudaErrors(hipMemsetAsync(m_device_eigenvectors, 0, 4 * 4 * sizeof(double), m_stream));
    build_matrix_F_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(m_device_atom_positions, m_device_reference_positions, m_device_eigenvectors, m_num_atoms, d_count);

    nvtxRangePushEx(&mEventAttrib);
#endif // defined(USE_CUDA_GRAPH)
#if defined(USE_NR)
#if defined(USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Jacobi node
        hipGraphNode_t jacobi4x4Node;
        hipKernelNodeParams kernelNodeParams = {0};
        const void *kernelArgs[] = {
            &m_device_eigenvectors, &m_device_eigenvalues, &max_reached};
        kernelNodeParams.func           = (void*)jacobi_4x4;
        kernelNodeParams.gridDim        = dim3(1, 1, 1);
        kernelNodeParams.blockDim       = dim3(2, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   =
            const_cast<void**>(kernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&jacobi4x4Node, m_graph, &last_node, 1, &kernelNodeParams));
        last_node = jacobi4x4Node;
    }
    // getLastCudaError("Line 312\n");
#else
    jacobi_4x4<<<1,2,0,m_stream>>>(m_device_eigenvectors, m_device_eigenvalues, max_reached);
    if (max_reached[0] > 0) {
        std::cerr << "Maximum number of iterations reached!\n";
    }
#endif // defined(USE_CUDA_GRAPH)
#else
    const size_t n_cols = 4;
    cusolver_status = hipsolverDnDsyevj(cusolverH, jobz, uplo, n_cols, m_device_eigenvectors, n_cols, m_device_eigenvalues, device_work, lwork, devInfo, syevj_info);
#endif
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Build rotation matrix node
        hipGraphNode_t buildRotationMatrixKernelNode;
        hipKernelNodeParams kernelNodeParams = {0};
        size_t max_eigenvalue_index = 3;
        void *kernelArgs[] = {
            &m_device_eigenvectors, &m_device_rotation_matrix, &max_eigenvalue_index};
        kernelNodeParams.func           = (void*)build_rotation_matrix_kernel;
        kernelNodeParams.gridDim        = dim3(1, 1, 1);
        kernelNodeParams.blockDim       = dim3(1, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = kernelArgs;
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(&buildRotationMatrixKernelNode, m_graph, &last_node, 1, &kernelNodeParams));
        last_node = buildRotationMatrixKernelNode;
    }
#else
    nvtxRangePop();
    // hipStreamSynchronize(m_stream);
    // build the optimal rotation matrix
    build_rotation_matrix_kernel<<<1,1,0,m_stream>>>(m_device_eigenvectors, m_device_rotation_matrix);
    // hipStreamSynchronize(m_stream);
#endif
}

// compute the optimal rmsd
#if defined (USE_CUDA_GRAPH)
double OptimalRotation::minimalRMSD() {
#else
double OptimalRotation::minimalRMSD() const {
#endif
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
#if defined (USE_CUDA_GRAPH)
    if (graphCreated == false) {
        // Memsets
        hipGraphNode_t counterSetNode, deviceRMSDSetNode;
        hipMemsetParams memsetParams = {0};
        memsetParams.dst            = d_count;
        memsetParams.value          = 0;
        memsetParams.elementSize    = 1 * sizeof(unsigned int);
        memsetParams.width          = 1;
        memsetParams.height         = 1;
        checkCudaErrors(hipGraphAddMemsetNode(
            &counterSetNode, m_graph, &last_node, 1, &memsetParams));
        // last_node = counterSetNode;
        memsetParams.dst            = m_device_rmsd;
        memsetParams.elementSize    = sizeof(float);
        memsetParams.width          = sizeof(double) / memsetParams.elementSize;
        checkCudaErrors(hipGraphAddMemsetNode(
            &deviceRMSDSetNode, m_graph, &last_node, 1, &memsetParams));
        // last_node = deviceRMSDSetNode;
        hipGraphNode_t RMSDDependencies[] = {counterSetNode, deviceRMSDSetNode};
        // Compute RMSD
        hipGraphNode_t RMSDKernelNode;
        hipKernelNodeParams kernelNodeParams = {0};
        const size_t max_eigenvalue_index = 3;
        const void *kernelArgs[] = {
            &m_device_atom_positions,
            &m_device_reference_positions,
            &m_device_eigenvalues,
            &m_device_rotation_matrix,
            &m_device_rmsd, &m_num_atoms, &d_count,
            &max_eigenvalue_index};
        kernelNodeParams.func           =
            (void*)compute_optimal_rmsd_kernel<block_size>;
        kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
        kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams   = const_cast<void**>(kernelArgs);
        kernelNodeParams.extra          = NULL;
        checkCudaErrors(hipGraphAddKernelNode(
            &RMSDKernelNode, m_graph, RMSDDependencies, 2, &kernelNodeParams));
        last_node = RMSDKernelNode;
        // Instantiate graph
        checkCudaErrors(hipGraphInstantiate(&m_instance, m_graph, NULL, NULL, 0));
        graphCreated = true;
        hipGraphDebugDotFlags dotFlags = hipGraphDebugDotFlagsVerbose;
        checkCudaErrors(hipGraphDebugDotPrint(m_graph, "graph.dot", dotFlags));
    }
    // Run graph
    checkCudaErrors(hipGraphLaunch(m_instance, m_stream));
#else
    checkCudaErrors(hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    compute_optimal_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(
        m_device_atom_positions,
        m_device_reference_positions,
        m_device_eigenvalues,
        m_device_rotation_matrix,
        m_device_rmsd, m_num_atoms, d_count);
#endif
    checkCudaErrors(hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream));
    checkCudaErrors(hipStreamSynchronize(m_stream));
    return *(m_host_rmsd);
}

#if defined (USE_CUDA_GRAPH)
void OptimalRotation::resetGraph() {
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipGraphExecDestroy(m_instance));
    checkCudaErrors(hipGraphDestroy(m_graph));
    checkCudaErrors(hipStreamDestroy(m_stream));
    graphCreated = false;
    last_node = NULL;
    // Recreate graph
    checkCudaErrors(hipStreamCreate(&m_stream));
    checkCudaErrors(hipGraphCreate(&m_graph, 0));
}
#endif // USE_CUDA_GRAPH

#if defined (USE_CUDA_GRAPH)
#else
// compute the optimal rmsd with respect to a specified frame
double OptimalRotation::minimalRMSD(const host_vector<AtomPosition>& atom_positions) const {
    checkCudaErrors(hipMemsetAsync(m_device_rmsd, 0, 1 * sizeof(double), m_stream));
    AtomPosition* device_atom_positions;
    checkCudaErrors(hipMalloc(&device_atom_positions, m_num_atoms * sizeof(AtomPosition)));
    const int num_blocks = (m_num_atoms + block_size - 1) / block_size;
    // copy data to device
    checkCudaErrors(hipMemcpyAsync(device_atom_positions, atom_positions.data(), m_num_atoms * sizeof(AtomPosition), hipMemcpyHostToDevice, m_stream));
    // compute geometric center
    checkCudaErrors(hipMemsetAsync(m_center_tmp, 0, sizeof(double3), m_stream));
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    get_center_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms, d_count);
    move_atom_to_center_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_center_tmp, m_num_atoms);
    // we assume the reference frame is already moved to its center of geometry
    // rotate the atoms
    rotate_atoms_kernel<<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_rotation_matrix, m_num_atoms);
    // compute rmsd directly
    checkCudaErrors(hipMemsetAsync(d_count, 0, 1 * sizeof(unsigned int), m_stream));
    compute_rmsd_kernel<block_size><<<num_blocks, block_size, 0, m_stream>>>(device_atom_positions, m_device_reference_positions, m_device_rmsd, m_num_atoms, d_count);
    checkCudaErrors(hipMemcpyAsync(m_host_rmsd, m_device_rmsd, 1 * sizeof(double), hipMemcpyDeviceToHost, m_stream));
//     hipFree(device_rmsd);
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipFree(device_atom_positions));
    return *(m_host_rmsd);
}
#endif

OptimalRotation::~OptimalRotation() {
    checkCudaErrors(hipStreamSynchronize(m_stream));
    checkCudaErrors(hipFree(m_device_atom_positions));
    checkCudaErrors(hipFree(m_device_reference_positions));
    checkCudaErrors(hipFree(m_device_rotation_matrix));
    checkCudaErrors(hipFree(m_device_eigenvalues));
    checkCudaErrors(hipFree(m_device_eigenvectors));
    checkCudaErrors(hipStreamDestroy(m_stream));
#if !defined (USE_NR)
    hipsolverDnDestroySyevjInfo(syevj_info);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    checkCudaErrors(hipFree(devInfo));
    checkCudaErrors(hipFree(device_work));
#endif
    checkCudaErrors(hipFree(m_center_tmp));
    checkCudaErrors(hipFree(m_device_rmsd));
    checkCudaErrors(hipHostFree(m_host_rmsd));
    checkCudaErrors(hipFree(d_count));
#if defined(USE_NR)
    checkCudaErrors(hipHostFree(max_reached));
#endif
#if defined (USE_CUDA_GRAPH)
    if (m_graph) checkCudaErrors(hipGraphDestroy(m_graph));
    if (m_instance) checkCudaErrors(hipGraphExecDestroy(m_instance));
#endif
}
